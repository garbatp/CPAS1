#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_complex.h"
#include "math_functions.h"
#include "hip/hip_math_constants.h":
#include <hipfft/hipfft.h>

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

struct kernelConf
{
	dim3 block;
	dim3 grid;
};

kernelConf* conf_FFT_Shift(int N, int batch)
{
	kernelConf* conf = (kernelConf*)malloc(sizeof(kernelConf));

	int threadsPerBlock_X;

	threadsPerBlock_X = 1024;

	conf->block = dim3(threadsPerBlock_X, 1, 1);
	conf->grid = dim3(((N*batch / threadsPerBlock_X)) + 1, 1, 1);

	return conf;
}

__global__ void cufftShift_2D(hipfftComplex* data, int N, int batch)
{

	int sLine = N;
	int sSlice = N * N;


	int sEq1 = (sSlice + sLine) / 2;
	int sEq2 = (sSlice - sLine) / 2;

	int threadIdxX = threadIdx.x;
	int blockDimX = blockDim.x;
	int blockIdxX = blockIdx.x;

	hipfftComplex regTemp;
	int index = ((blockIdxX * blockDimX) + threadIdxX);
	int batchNumber = index / (N*N);

	int yIndex = (index / N) - batchNumber*N;
	int xIndex = index - (N*(index / N));


	if (batchNumber <= (batch - 1) && xIndex < N / 2)
	{
		if (batchNumber <= (batch - 1) && yIndex < N / 2)
		{
			regTemp = data[index];


			data[index] = data[index + sEq1];


			data[index + sEq1] = regTemp;
		}
	}
	else
	{
		if (batchNumber <= (batch - 1) && yIndex < N / 2)
		{
			regTemp = data[index];

			data[index] = data[index + sEq2];


			data[index + sEq2] = regTemp;
		}
	}


}

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__device__ __forceinline__ hipComplex expf(hipComplex z)
{

	hipComplex res;
	float t = expf(z.x);
	sincosf(z.y, &res.y, &res.x);
	res.x *= t;
	res.y *= t;

	return res;

}

__global__ void calculate(hipComplex *fths, int *xo, int *yo, double *uo, float *zo2, float dfxs, float lambda, float k0, int Ts, float *fxs, float * y0seg, float* x0seg, int S_Bx, int S_By, int N_Bx, int N_By)
{

	//	int	index = blockIdx.y*(blockDim.x*gridDim.x) + blockIdx.x*blockDim.x + threadIdx.x;
	//	int index = threadIdx.x*blockIdx.z + blockIdx.y*blockDim.z + blockIdx.x*blockDim.x;


	//	int blockId = blockIdx.x + blockIdx.y * gridDim.x  + gridDim.x * gridDim.y * blockIdx.z;
	//	int threadId = blockId * blockDim.x + threadIdx.x;

	//	int pt_indx = threadIdx.x*blockIdx.z;


	float yp = yo[threadIdx.x] - y0seg[blockIdx.y];

	float xp = xo[threadIdx.x] - x0seg[blockIdx.x];

	float rp = sqrt(zo2[threadIdx.x] + xp*xp + yp*yp);


	float inv_rp = 1 / rp;

	float fxp = xp*inv_rp / lambda;
	float fyp = yp*inv_rp / lambda;



	int iifx = round(fxp / dfxs) + S_Bx / 2 + 1;
	int iify = round(fyp / dfxs) + S_By / 2 + 1;

	if (iifx <= 0 || iifx > S_Bx || iify <= 0 || iify > S_Bx){
		iifx = S_Bx / 2 + 1;
		iify = S_Bx / 2 + 1;
	}



	hipComplex c0;
	hipComplex arg;
	arg.x = (k0*rp - 2 * HIP_PI_F*(fxs[iifx] + fxs[iify])*(Ts / 2)*inv_rp);

	c0 = expf(arg);
	//	c0.x = uo[blockDim.x] * c0.x;
	//	c0.y = uo[blockDim.x] * c0.y;

	//fths[threadId] = c0;
	//	Nep[threadId] = iifx;
	//	Nip[threadId] = iify;

	fths[iifx + blockIdx.x*S_Bx + iify*S_Bx*N_Bx + blockIdx.x* S_Bx*N_Bx*S_By].x += c0.x;
	fths[iifx + blockIdx.x*S_Bx + iify*S_Bx*N_Bx + blockIdx.x* S_Bx*N_Bx*S_By].y += c0.y;
}

hipfftResult preparePlan2D(hipfftHandle* plan, int nRows, int nCols, int batch){

	int n[2] = { nRows, nCols };

	hipfftResult result = hipfftPlanMany(plan,
		2, //rank
		n, //dimensions = {nRows, nCols}
		0, //inembed
		batch, //istride
		1, //idist
		0, //onembed
		batch, //ostride
		1, //odist
		HIPFFT_C2C, //hipfftType
		batch /*batch*/);

	if (result != 0){

		//		std::cout << "preparePlan2D error, result: " << result << "/n";
		return result;
	}
	return result;
}

hipfftResult execute2D(hipfftHandle* plan, hipfftComplex* idata, hipfftComplex* odata, int direction){

	hipfftResult result = hipfftExecC2C(*plan, idata, odata, direction);

	if (result != 0){

		//		cout << "execute2D error, result: " << result << "/n";
		return result;
	}
	return result;
}


__global__ void copy2bitmap(hipComplex *H, int *bitmap_H)
{

}


__global__ void asemble(hipComplex *fths, int *xo, int *yo, int *uo, float *zo2, float *dfxs, int *Nxs, float *lambda, int *Ts, float * fxs, float* y0seg, float* x0seg, int* Nep, int* Nip)
{
	//int	index = blockIdx.y*(blockDim.x*gridDim.x) + blockIdx.x*blockDim.x + threadIdx.x;
	//	int index = threadIdx.x*blockIdx.z + blockIdx.y*blockDim.z + blockIdx.x*blockDim.x;


	int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int threadId = blockId * blockDim.x + threadIdx.x;

	int pt_indx = threadIdx.x*blockIdx.z;

	//	fths2[] = fths2[] + fths[threadId]

	//		fths[threadId] = c0;
	//	Nep[threadId] = iifx;
	//	Nip[threadId] = iify;


}


void CPAS_CGH_3DPS_2d(int Np, int* xo, int* yo, double* zo, double* uo, int Nx, int Ny, int dx, float lambda, int S_Bx, int S_By, hipComplex* fths_p)
{
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	double k0 = 2 * HIP_PI_F / lambda;

	int x_size = (Nx / 2) + ((Nx / 2) - 1) + 1;
	int y_size = (Ny / 2) + ((Ny / 2) - 1) + 1;

	float *x = (float*) malloc(x_size * sizeof(float));
	float *y = (float*) malloc(y_size * sizeof(float));

	for (int t = 0; t < x_size; t++){
		x[t] = (-Nx / 2 + t)*dx;
	}

	for (int t = 0; t < y_size; t++){
		y[t] = (-Ny / 2 + t)*dx;
	}

	int N_Bx = Nx / S_Bx; // doda� obs�ug� nie ca�kowitych dziele�
	int N_By = Ny / S_By;

	int Ts = S_Bx*dx;

	float dfxs = 1 / (float)Ts;

	int fxs_size = (S_Bx / 2) + ((S_Bx / 2) - 1) + 1;
	float *fxs = (float*)malloc(fxs_size * sizeof(float));

	for (int t = 0; t < fxs_size; t++){
		fxs[t] = (float)(-S_Bx / 2 + t)*dfxs;
	}

	float * x0seg = (float*)malloc((N_Bx)* sizeof(float));

	for (int t = 0; t < N_By; t++){
		x0seg[t] = (x[0] + (t*Ts) + Ts / 2);
	}

	float * y0seg = (float*)malloc((N_By)* sizeof(float));

	for (int t = 0; t < N_By; t++){
		y0seg[t] = (y[0] + (t*Ts) + Ts / 2);
	}
	/*	float * nseg_bx = (float*)malloc((Nosx)* sizeof(float));

	for (int t = 0; t < Nosx; t++){
	nseg_bx[t] = (1 + (t*Nxs));

	}

	float * nseg_by = (float*)malloc((Nosy)* sizeof(float));

	for (int t = 0; t < Nosy; t++){
	nseg_by[t] = (1 + (t*Nxs));
	}

	float *h = (float*)calloc(Nx, sizeof(float));
	*/

	float *z02;
	z02 = (float*)malloc((Np)* sizeof(float));

	for (int t = 0; t < Np; t++)
		z02[t] = zo[t] * zo[t];



	//	hipMalloc(&fths_p, sizeof(hipComplex)*N_Bx*N_By*S_Bx*S_By);
	//	hipMemset(fths_p, 0, sizeof(hipComplex)*N_Bx*N_By*S_Bx*S_By);


	int *d_xo;
	int *d_yo;
	float *d_z02;

	hipMalloc((void**)&d_xo, sizeof(int)*Np);
	hipMalloc((void**)&d_yo, sizeof(int)*Np);
	hipMalloc((void**)&d_z02, sizeof(float)*Np);

	hipMemcpy(d_xo, xo, Np*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_yo, yo, Np*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_z02, z02, Np*sizeof(float), hipMemcpyHostToDevice);

	float *d_fxs;
	float *d_y0seg;
	float *d_x0seg;

	hipMalloc((void**)&d_x0seg, sizeof(float)*N_Bx);
	hipMalloc((void**)&d_y0seg, sizeof(float)*N_By);
	hipMalloc((void**)&d_fxs, sizeof(float)*fxs_size);

	hipMemcpy(d_fxs, fxs, fxs_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x0seg, x0seg, N_Bx*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y0seg, y0seg, N_By*sizeof(float), hipMemcpyHostToDevice);

	dim3 grid;
	grid.x = N_Bx;//y
	grid.y = N_By;//x

	dim3 block;
	block.x = Np; //z
	block.y = 1;

	hipEventRecord(start, 0);
	calculate << < grid, block >> >(fths_p, d_xo, d_yo, uo, d_z02, dfxs, lambda, k0, Ts, d_fxs, d_y0seg, d_x0seg, S_Bx, S_Bx, N_Bx, N_By);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);


	/*	hipComplex *host;
	host = (hipComplex*)malloc(sizeof(hipComplex)*Nosx*Nosy*Np);
	hipMemcpy(host, fths, sizeof(hipComplex)*Nosx*Nosy*Np, hipMemcpyDeviceToHost);
	*/
}

int main()
{

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*START CUDA CALC PART - DEKLARACJE*/
	int Nx = 512;
	int Ny = 512;
	int dx = 8;
	float lambda = 0.5;
	/*START CUDA FFT 2D PART - DEKLARACJE*/
	int S_Bx = 16;
	int S_By = 16;

	hipfftComplex* h_out; //dane wynikowe CPU
	hipfftComplex* holo; //dane wyj�ciowe GPU

	int batch = Nx / S_Bx * Ny / S_By;  //N_Bx*N_By
	hipfftHandle forwardPlan;

	preparePlan2D(&forwardPlan, S_Bx, S_By, batch);

	h_out = (hipfftComplex*)malloc(sizeof(hipfftComplex) *S_Bx*S_By*batch); //allokacja pami�ci na wynik (CPU)

	hipMalloc(&holo, sizeof(hipfftComplex)*S_Bx*S_By*batch); //allokacja pami�ci na dane wyj�ciowe (GPU)
	hipMemset(holo, 0, sizeof(hipfftComplex)*S_Bx*S_By*batch); //Wype�nianie zaalokowanej pami�ci zerami (GPU)

	/*END CUDA FFT 2D PART - DEKLARACJE*/

	/*Kod kernela*/
	int Np = 1024;

	int *xo;
	int *yo;
	double *zo;
	double *uo;

	xo = (int*)malloc((Np)* sizeof(int));
	yo = (int*)malloc((Np)* sizeof(int));
	zo = (double*)malloc((Np)* sizeof(double));
	uo = (double*)malloc((Np)* sizeof(double));

	for (int tt = 0; tt < Np; tt++)
	{
		xo[tt] = tt;
		yo[tt] = tt;
		zo[tt] = tt;
	}

	double W = 0.1e3;
	double dxo = W / 10.0;
	int foo = 0;

	for (foo = 0; foo < Np; foo++)
	{
		xo[foo] = W;
		yo[foo] = W;
		zo[foo] = 50e3; // 5e3;
		uo[foo] = 3.14;
	}



	hipComplex *fths_p;

	hipfftComplex* fhs;

	//	hipMalloc(&fhs, sizeof(hipfftComplex)*S_Bx*S_By*batch); //allokacja pami�ci na dane wej�ciowe (GPU)
	hipMalloc(&fths_p, sizeof(hipComplex)*Nx*Ny);
	hipMemset(fths_p, 0, sizeof(hipComplex)*Nx*Ny);

	hipEventRecord(start, 0);
	/*START CUDA CALC PART */
	CPAS_CGH_3DPS_2d(Np, xo, yo, zo, uo, Nx, Ny, dx, lambda, S_Bx, S_By, fths_p);

	/*END CUDA CALC PART */
	hipMemcpy(h_out, holo, sizeof(hipfftComplex)*S_Bx*S_By*batch, hipMemcpyDeviceToHost);

	for (int iii = 0; iii < Nx*Ny; iii++)
	{
		if (h_out[iii].x != 0)
			printf("T: %f + i%f\n", 10e15*h_out[iii].x, 10e15*h_out[iii].y);
	}


	FILE *fp_real;
	FILE *fp_imag;

	if ((fp_real = fopen("fths_p_real.txt", "w")) != NULL && (fp_imag = fopen("fths_p_imag.txt", "w")) != NULL) {

		for (int iii = 0; iii < S_Bx*S_By*batch; iii++)
		{
			fprintf(fp_real, "%f \n", h_out[iii].x);
			fprintf(fp_imag, "%f \n", h_out[iii].y);
		}
	}

	fclose(fp_real);
	fclose(fp_imag);

	/*START CUDA FFT_SHIFT PART */
	kernelConf * conf = conf_FFT_Shift(S_Bx*S_By, batch);

	cufftShift_2D << <conf->grid, conf->block >> >(fths_p, S_Bx, batch);
	/*END CUDA FFT_SHIFT PART */

	/*START CUDA FFT PART */
	execute2D(&forwardPlan, fths_p, holo, HIPFFT_FORWARD);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	/*Wyswietlanie modulu/fazy*/


	/*END CUDA FFT PART */



	// Retrieve result from device and store it in host array
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);
	printf("Time for the kernel: %f ms\n", time);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	/*	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipDeviceReset failed!");
	return 1;
	}
	*/
	return 0;
}

